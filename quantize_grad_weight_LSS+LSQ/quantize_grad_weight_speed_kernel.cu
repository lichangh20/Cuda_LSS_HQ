#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <cmath>
#include <iostream>
#include <sstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <ctime>
#include "cuda_fp16.hpp"
#include "hip/hip_fp16.h"

#include "hip/hip_runtime.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include <hiprand/hiprand_kernel.h>
#include <tuple>
#include <bits/stdc++.h>
// #include <torch/distributions/gumbel.h>

#include "torch/script.h"
using namespace torch::indexing;

template<typename scalar_t>
__global__ void quantize_cuda_kernel(const scalar_t * __restrict__  MatI, int8_t * first_transform, scalar_t * __restrict__  first_quantize, int8_t * second_transform, scalar_t * __restrict__  second_quantize, 
                                    const int num_bins_half, const int num_bins_clamp, const float scale, int size, unsigned long seed){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x<size){
        // set random value
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(seed, x, 0, &state);
        const float noise = hiprand_uniform(&state);

        float trans_input = MatI[x] * scale;

        float tmp1 = round(trans_input / num_bins_half);
        int firstTransform = std::clamp((int)(tmp1), -num_bins_clamp, num_bins_clamp);
        first_transform[x] = firstTransform;
        // float quantize = (transform + 8) / scale + zero_point;
        first_quantize[x] = firstTransform * num_bins_half / scale;

        float tmp2 = round(trans_input - firstTransform * num_bins_half + noise - 0.5);
        int secondTransform = std::clamp((int)(tmp2), -num_bins_clamp, num_bins_clamp);
        second_transform[x] = secondTransform;
        second_quantize[x] = secondTransform / scale;
    }
}

// template<typename scalar_t>
// __global__ void second_quantize_cuda_kernel(const scalar_t * __restrict__  MatI, int8_t * MatO_transform, scalar_t * __restrict__  MatO_quantize, const float scale, const float  zero_point, int size, unsigned long seed){
//     int x = threadIdx.x + blockIdx.x * blockDim.x;
//     if (x<size){
//         // set random value
//         hiprandStatePhilox4_32_10_t state;
//         hiprand_init(seed, x, 0, &state);
//         const float noise = hiprand_uniform(&state);

//         float input = MatI[x];
//         // scalar_t tmp1 = (input - zero_point) * scale + noise - 8.5;
//         // // scalar_t tmp1 = (MatI[x] - zero_point) * scale - 8;
//         // int tmp2 = tmp1;
//         // int bias = (tmp1 - tmp2) * 2;
//         // MatO_transform[x] = std::clamp(tmp2+bias, -8, 7);
//         float tmp1 = round((input - zero_point) * scale + noise - 8.5);
//         MatO_transform[x] = std::clamp((int)(tmp1), -8, 7);
//         MatO_quantize[x] = (MatO_transform[x] + 8) / scale + zero_point;
//     }
// }

__global__ void pack_cuda_kernel(int8_t * in, int8_t * out, int size){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x<size){
        out[x] = (in[(x<<1)+1] << 4) | (in[x<<1] & 15);
    }
}

hipError_t CutlassSgemmNN_fp16(
  const int M,
  const int N,
  const int K,
  const cutlass::half_t *A,
  int lda,
  const cutlass::half_t *B,
  int ldb,
  cutlass::half_t *C,
  int ldc) {

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = float;                 // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = cutlass::half_t;                       // <- data type of elements in input matrix A
using ElementInputB = cutlass::half_t;                       // <- data type of elements in input matrix B
using ElementOutput = cutlass::half_t;                      // <- data type of elements in output matrix D

// The code section below describes matrix layout of input and output matrices. Column Major for
// Matrix A, Row Major for Matrix B and Row Major for Matrix C
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::RowMajor;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm70;

// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock =
    cutlass::gemm::GemmShape<256, 128, 32>;  // <- threadblock tile M = 128, N = 256, K = 64
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 64 
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<8, 8, 4>;  // <- MMA Op tile M = 8, N = 8, K = 16

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>;  // <- ??

// This code section describes the epilogue part of the kernel
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // <- data type of output matrix
    4,  // <- the number of elements per vectorized
                                                       // memory access. For a byte, it's 16
                                                       // elements. This becomes the vector width of
                                                       // math instructions in the epilogue too
    ElementAccumulator,                                // <- data type of accumulator
    ElementComputeEpilogue>;  // <- data type for alpha/beta in linear combination function

// Number of pipelines you want to use
constexpr int NumStages = 2;

using Gemm = cutlass::gemm::device::Gemm<ElementInputA,
                                         LayoutInputA,
                                         ElementInputB,
                                         LayoutInputB,
                                         ElementOutput,
                                         LayoutOutput,
                                         ElementAccumulator,
                                         MMAOp,
                                         SmArch,
                                         ShapeMMAThreadBlock,
                                         ShapeMMAWarp,
                                         ShapeMMAOp,
                                         EpilogueOp,
                                         SwizzleThreadBlock,
                                         NumStages>;
  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);
    
  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename Gemm::Arguments arguments{problem_size,  // <- problem size of matrix multiplication
                                     {A, lda},  // <- reference to matrix A on device
                                     {B, ldb},  // <- reference to matrix B on device
                                     {C, ldc},  // <- reference to matrix C on device
                                     {C, ldc},  // <- reference to matrix D on device
                                     {alpha, beta},          // <- tuple of alpha and beta
                                     split_k_slices};        // <- k-dimension split factor
    Gemm gemm_op;

    // Launch initialized CUTLASS kernel
    cutlass::Status status = gemm_op(arguments);

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  const int M,
  const int N,
  const int K,
  const cutlass::int4b_t *A,
  int lda,
  const cutlass::int4b_t *B,
  int ldb,
  int32_t *C,
  int ldc) {

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = int32_t;                 // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = cutlass::int4b_t;                       // <- data type of elements in input matrix A
using ElementInputB = cutlass::int4b_t;                       // <- data type of elements in input matrix B
using ElementOutput = int32_t;                      // <- data type of elements in output matrix D

// The code section below describes matrix layout of input and output matrices. Column Major for
// Matrix A, Row Major for Matrix B and Row Major for Matrix C
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::RowMajor;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm75;

// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock =
    cutlass::gemm::GemmShape<256, 128, 128>;  // <- threadblock tile M = 128, N = 256, K = 64
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 128>;  // <- warp tile M = 64, N = 64, K = 64 
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<8, 8, 32>;  // <- MMA Op tile M = 8, N = 8, K = 16

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>;  // <- ??

// This code section describes the epilogue part of the kernel
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // <- data type of output matrix
    4,  // <- the number of elements per vectorized
                                                       // memory access. For a byte, it's 16
                                                       // elements. This becomes the vector width of
                                                       // math instructions in the epilogue too
    ElementAccumulator,                                // <- data type of accumulator
    ElementComputeEpilogue>;  // <- data type for alpha/beta in linear combination function

// Number of pipelines you want to use
constexpr int NumStages = 2;

using Gemm = cutlass::gemm::device::Gemm<ElementInputA,
                                         LayoutInputA,
                                         ElementInputB,
                                         LayoutInputB,
                                         ElementOutput,
                                         LayoutOutput,
                                         ElementAccumulator,
                                         MMAOp,
                                         SmArch,
                                         ShapeMMAThreadBlock,
                                         ShapeMMAWarp,
                                         ShapeMMAOp,
                                         EpilogueOp,
                                         SwizzleThreadBlock,
                                         NumStages>;
  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);
    
  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename Gemm::Arguments arguments{problem_size,  // <- problem size of matrix multiplication
                                     {A, lda},  // <- reference to matrix A on device
                                     {B, ldb},  // <- reference to matrix B on device
                                     {C, ldc},  // <- reference to matrix C on device
                                     {C, ldc},  // <- reference to matrix D on device
                                     {alpha, beta},          // <- tuple of alpha and beta
                                     split_k_slices};        // <- k-dimension split factor
  
    Gemm gemm_op;
    cutlass::Status status = gemm_op(arguments);

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

#define N_THREADS 256

template<typename scalar_t>
__global__ void dequantize_cuda_kernel(const int32_t * gemm1, const int32_t * gemm2, const scalar_t * __restrict__ gemm3, 
                                        const scalar_t * __restrict__ gemm4, scalar_t * __restrict__ output, 
                                        const float scale_gemm1, const float scale_gemm2, int size){  
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    // int row = x / ny, col = x - row * ny;

    if (x<size){
       output[x] = gemm1[x] * scale_gemm1 + gemm2[x] * scale_gemm2 + gemm3[x] + gemm4[x];
        // output[x] = 0;
    }
}

// template<typename scalar_t>
// __global__ void dequantize_cuda_kernel_fp16(const scalar_t * __restrict__ gemm1, const scalar_t * __restrict__ gemm2, scalar_t * __restrict__ output, 
//                                         int size){  
//     int x = threadIdx.x + blockIdx.x * blockDim.x;

//     if (x<size){
//        output[x] = gemm1[x] + gemm2[x];
//     }
// }


__device__ __inline__ c10::Half __shfl_down_sync(const unsigned mask, const c10::Half var,
                                                 const unsigned int delta, const int width) {
  __half var_ = var;
  return __shfl_down_sync(mask, var_, delta, width);
}

//TODO: N means rows, D means cols
template<typename scalar_t>
__global__ void linalg_norm_cuda_kernel(const scalar_t * __restrict__ in, scalar_t * __restrict__ linalg, int N, int D, int stride_D){
  float sum_val = 0;

  for (int64_t k1_outer = 0; k1_outer < stride_D; ++k1_outer) {
    float temp = in[blockIdx.x * D + (k1_outer << 5) + threadIdx.x];
    sum_val += temp * temp;
  }

  unsigned int mask;
  float sum_val_t;
  mask = __activemask();

  sum_val_t = __shfl_down_sync(mask, sum_val, 16, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 8, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 4, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 2, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 1, 32);
  sum_val += sum_val_t;
  linalg[blockIdx.x] = sqrt(sum_val);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> quantize_cuda(torch::Tensor x, int num_bits, torch::Tensor y, torch::Tensor qy, float scaley, torch::Tensor hadamard_weight, torch::Tensor scale_weight){
    // std::vector<double> time_vector;
    int nz = x.size(0);
    int nx = x.size(1);
    int ny = y.size(1);

    // hipDeviceSynchronize();
    // clock_t time_quantize1_start = clock();

    auto option_transform = torch::TensorOptions().dtype(torch::kInt8).device(x.device());
    auto option_quantize = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    torch::Tensor first_transform = torch::empty({nz, nx}, option_transform);
    torch::Tensor first_quantize = torch::empty({nz, nx}, option_quantize);
    torch::Tensor second_transform = torch::empty({nz, nx}, option_transform);
    torch::Tensor second_quantize = torch::empty({nz, nx}, option_quantize);

    dim3 block(N_THREADS);
    dim3 grid1((nx*nz-1)/block.x+1);
    int size_quantize = nz * nx ;
    // process of first quantize
    float mn = std::min(x.min().item<float>() - 1e-8, 0.);
    float mx = std::max(x.max().item<float>() + 1e-8, 0.);

    int num_bins_half = pow(2, num_bits) - 2;
    int num_bins = num_bins_half * num_bins_half;
    int num_bins_clamp = num_bins_half / 2 - 1;

    float scale1 = num_bins / (2 * max(fabs(mn), fabs(mx)));

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "quantize_cuda", ([&] {
    quantize_cuda_kernel<scalar_t><<<grid1, block>>>(
        x.data_ptr<scalar_t>(),
        first_transform.data_ptr<int8_t>(),
        first_quantize.data_ptr<scalar_t>(),
        second_transform.data_ptr<int8_t>(),
        second_quantize.data_ptr<scalar_t>(),
        num_bins_half, num_bins_clamp,
        scale1, size_quantize,rand());
    }));

    // hipDeviceSynchronize();
    // clock_t time_quantize2_end = clock();

    // leverage score
    // TODO: use dim=0 because torch.linalg only supports dim=1
    int threads = 32;
    int blocks = nz;

    auto x1_len = torch::empty({nz,}, option_quantize);
    auto x2_len = torch::empty({nz,}, option_quantize);
    auto y_len = torch::empty({nz,}, option_quantize);

    int stride_x = nx / 32;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x1_len.scalar_type(), "linalg_cuda", ([&] {
    linalg_norm_cuda_kernel<scalar_t><<<blocks, threads>>>(
        first_quantize.data_ptr<scalar_t>(), 
        x1_len.data_ptr<scalar_t>(),
        nz,nx,stride_x);
    }));
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x2_len.scalar_type(), "linalg_cuda", ([&] {
    linalg_norm_cuda_kernel<scalar_t><<<blocks, threads>>>(
        second_quantize.data_ptr<scalar_t>(), 
        x2_len.data_ptr<scalar_t>(),
        nz,nx,stride_x);
    }));
    int stride_y = ny / 32;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(y.scalar_type(), "linalg_cuda", ([&] {
    linalg_norm_cuda_kernel<scalar_t><<<blocks, threads>>>(
        y.data_ptr<scalar_t>(), 
        y_len.data_ptr<scalar_t>(),
        nz,ny,stride_y);
    }));

    // TODO: whether need to change dtype from half into float? It depends 
    auto vec_norm = torch::cat({torch::mul(x1_len, y_len), torch::mul(x2_len, y_len)}).to(torch::kFloat32);
    // auto vec_norm = torch::cat({torch::mul(x1_len, y_len), torch::mul(x2_len, y_len)});
    int len_norm = vec_norm.numel();

    int cnt = 0;
    auto norm_weight_loop = vec_norm * len_norm / (2 * vec_norm.sum());
    bool whileloop = (norm_weight_loop.max() > 1).item<bool>();
    while (whileloop && cnt < len_norm / 2){
        auto small_index = torch::nonzero((norm_weight_loop < 1)).squeeze();
        auto small_value = norm_weight_loop.index({small_index});
        cnt = len_norm - small_index.numel();
        norm_weight_loop = torch::clamp(norm_weight_loop, 0, 1);
        bool breakloop = (small_value.max() == 0).item<bool>() && (small_value.min() == 0).item<bool>();
        if (breakloop)
            break;
        small_value = small_value * (len_norm / 2 - cnt) / small_value.sum();
        // norm_weight_loop[small_index] = small_value;
        norm_weight_loop.index_put_({small_index}, small_value);
        whileloop = (norm_weight_loop.max() > 1).item<bool>();
    } 
    auto sample_index = torch::bernoulli(norm_weight_loop);
    auto small_indices = torch::nonzero(sample_index.index({Slice({None, len_norm/2})}) == 1).squeeze(1);
    auto small_num = (norm_weight_loop.index({small_indices}) == 1).sum();
    // TODO: test if .int() can work in libtorch
    small_num = ((small_num / 32).floor() * 32).to(torch::kInt32);
    int small_num_ = small_num.item<int>();
    auto small_int_indices = small_indices.index({torch::nonzero(norm_weight_loop.index({small_indices}) == 1).squeeze(1)}).index({Slice({None, small_num_})});
    auto small_left_indices = small_indices.index({~torch::isin(small_indices, small_int_indices)});
    auto large_indices = torch::nonzero(sample_index.index({Slice(len_norm/2)}) == 1).squeeze(1);
    auto large_num = (norm_weight_loop.index({large_indices + len_norm / 2}) == 1).sum();
    // TODO: test if .int() can work in libtorch
    large_num = ((large_num / 32).floor() * 32).to(torch::kInt32);
    int large_num_ = large_num.item<int>();
    auto large_int_indices = large_indices.index({torch::nonzero(norm_weight_loop.index({large_indices + len_norm / 2}) == 1).squeeze(1)}).index({Slice({None, large_num_})});
    auto large_left_indices = large_indices.index({~torch::isin(large_indices, large_int_indices)});
    // auto _index = torch::nonzero((sample_index == 1)).squeeze();

    norm_weight_loop.index_put_({norm_weight_loop == 0}, 1e-10);

    
    // auto small_num_ = (_index < len_norm / 2).sum();
    // auto large_num_ = _index.numel() - small_num_;
    // auto small_indices = _index.index({Slice({None, small_num_.item<int>()})});
    // auto large_indices = _index.index({Slice(small_num_.item<int>())}) - int(len_norm / 2);
    // auto norm_weight_small = norm_weight_loop.index({small_indices});
    // auto norm_weight_large = norm_weight_loop.index({large_indices + len_norm / 2});
    // auto output = torch::cat({first_quantize, second_quantize});
    // output = output / norm_weight_loop.unsqueeze(1);

    // hipDeviceSynchronize();
    // clock_t time_leverage_end = clock();

    //TODO: suppose an easy situation so that it can be faster
    auto sample_x1 = first_transform.index({small_int_indices}).t().contiguous();
    auto sample_x2 = second_transform.index({large_int_indices}).t().contiguous();
    auto sample_y1 = qy.index({small_int_indices}).t().contiguous();
    auto sample_y2 = qy.index({large_int_indices}).t().contiguous();
    auto sample_x3 = (first_quantize.index({small_left_indices}).t() / norm_weight_loop.index({small_left_indices})).to(torch::kFloat16);
    auto sample_x4 = (second_quantize.index({large_left_indices}).t() / norm_weight_loop.index({large_left_indices + len_norm / 2})).to(torch::kFloat16);
    //todo:currently multiply a scaley to convert it into fp16
    auto sample_y3 = (qy.index({small_left_indices}) * scaley).to(torch::kFloat16);
    auto sample_y4 = (qy.index({large_left_indices}) * scaley).to(torch::kFloat16);
    // auto sample_x3 = (first_quantize.index({small_left_indices}).t().contiguous() / norm_weight_loop.index({small_left_indices})).to(torch::kFloat16);
    // auto sample_x4 = (second_quantize.index({large_left_indices}).t().contiguous() / norm_weight_loop.index({large_left_indices + len_norm / 2})).to(torch::kFloat16);
    // //todo:currently multiply a scaley to convert it into fp16
    // auto sample_y3 = (qy.index({small_left_indices}) * scaley).to(torch::kFloat16).t().contiguous();
    // auto sample_y4 = (qy.index({large_left_indices}) * scaley).to(torch::kFloat16).t().contiguous();

    // auto dequantize_sample_y = torch::cat({sample_y1, sample_y2}, 1);

    // hipDeviceSynchronize();
    // clock_t time_sample_end = clock();

    // pack process
    // auto option_transform = torch::TensorOptions().dtype(torch::kInt8).device(x.device());
    auto sample_x1_int4 = torch::empty({nx, small_num_>>1}, option_transform);
    auto sample_x2_int4 = torch::empty({nx, large_num_>>1}, option_transform);
    auto sample_y1_int4 = torch::empty({ny, small_num_>>1}, option_transform);
    auto sample_y2_int4 = torch::empty({ny, large_num_>>1}, option_transform);
    int grid_size_x1 = nx*small_num_/2;
    int grid_size_x2 = nx*large_num_/2;
    int grid_size_y1 = ny*small_num_/2;
    int grid_size_y2 = ny*large_num_/2;
    dim3 grid_pack_x1((grid_size_x1-1)/block.x+1);
    dim3 grid_pack_x2((grid_size_x2-1)/block.x+1);
    dim3 grid_pack_y1((grid_size_y1-1)/block.x+1);
    dim3 grid_pack_y2((grid_size_y2-1)/block.x+1);
    if (small_num_ > 0) {
        pack_cuda_kernel<<<grid_pack_x1,block>>>(sample_x1.data_ptr<int8_t>(), sample_x1_int4.data_ptr<int8_t>(), grid_size_x1);
        pack_cuda_kernel<<<grid_pack_y1,block>>>(sample_y1.data_ptr<int8_t>(), sample_y1_int4.data_ptr<int8_t>(), grid_size_y1);
    }
    if (large_num_ > 0) {
        pack_cuda_kernel<<<grid_pack_x2,block>>>(sample_x2.data_ptr<int8_t>(), sample_x2_int4.data_ptr<int8_t>(), grid_size_x2);
        pack_cuda_kernel<<<grid_pack_y2,block>>>(sample_y2.data_ptr<int8_t>(), sample_y2_int4.data_ptr<int8_t>(), grid_size_y2);
    }

    // hipDeviceSynchronize();
    // clock_t time_pack_end = clock();

    // gemm process
    hipError_t result;
    int lda_first = small_num_;
    int ldb_first = small_num_;
    int ldc = ny;
    // Chunked matrix multiplication
    auto gemm1 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
    if (small_num_ > 0) {
        result = CutlassSgemmNN(nx, ny, small_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x1_int4.data_ptr<int8_t>()), lda_first, reinterpret_cast<cutlass::int4b_t *>(sample_y1_int4.data_ptr<int8_t>()), ldb_first, gemm1.data_ptr<int32_t>(), ldc);
    } else {
        gemm1 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
    }
    // result = CutlassSgemmNN(nx, ny, small_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x1_int4.data_ptr<int8_t>()), lda_first, reinterpret_cast<cutlass::int4b_t *>(sample_y1_int4.data_ptr<int8_t>()), ldb_first, gemm1.data_ptr<int32_t>(), ldc);

    int lda_second = large_num_;
    int ldb_second = large_num_;
    auto gemm2 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
    if (large_num_ > 0) {
        result = CutlassSgemmNN(nx, ny, large_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x2_int4.data_ptr<int8_t>()), lda_second, reinterpret_cast<cutlass::int4b_t *>(sample_y2_int4.data_ptr<int8_t>()), ldb_second, gemm2.data_ptr<int32_t>(), ldc);
    } else {
        gemm2 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
    }
    // result = CutlassSgemmNN(nx, ny, large_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x2_int4.data_ptr<int8_t>()), lda_second, reinterpret_cast<cutlass::int4b_t *>(sample_y2_int4.data_ptr<int8_t>()), ldb_second, gemm2.data_ptr<int32_t>(), ldc);

    auto gemm3 = torch::matmul(sample_x3, sample_y3);
    auto gemm4 = torch::matmul(sample_x4, sample_y4);
    // auto gemm3 = torch::empty({nx, ny}, at::device(at::kCUDA).dtype(torch::kFloat16));
    // auto gemm4 = torch::empty({nx, ny}, at::device(at::kCUDA).dtype(torch::kFloat16));


    // auto gemm3 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kFloat16));

    // hipDeviceSynchronize();
    // clock_t time_gemm1_end = clock();

    // auto gemm2 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kFloat16));

    // hipDeviceSynchronize();
    // clock_t time_gemm2_end = clock();

    // dequantize process
    dim3 grid2((nx*ny-1)/block.x+1);
    // First dequantize higher 4 bits
    auto option_output = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    // auto sum_y1_column = torch::sum(qy.index({small_int_indices}), 0);
    // auto sum_y2_column = torch::sum(qy.index({large_int_indices}), 0);
    auto grad_output = torch::empty({nx,ny}, option_output);

    // float const_x1 = (8.0 / scale1 + zero_point1) * scaley;
    // float const_x2 = (8.0 / scale2 + zero_point2) * scaley;
    // float const_x = const_x1 + const_x2;
    float scale_gemm1 = scaley * num_bins_half / (scale1);
    float scale_gemm2 = scaley / (scale1);
    int size = nx*ny;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "dequantize_cuda", ([&] {
    dequantize_cuda_kernel<scalar_t><<<grid2, block>>>(
        gemm1.data_ptr<int32_t>(), 
        gemm2.data_ptr<int32_t>(),
        gemm3.data_ptr<scalar_t>(), 
        gemm4.data_ptr<scalar_t>(),
        grad_output.data_ptr<scalar_t>(),
        // sum_y1_column.data_ptr<int64_t>(),
        // sum_y2_column.data_ptr<int64_t>(),
        // const_x1, const_x2, 
        scale_gemm1, scale_gemm2,
        size);
    }));



    // AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "dequantize_cuda", ([&] {
    // dequantize_cuda_kernel_fp16<scalar_t><<<grid2, block>>>(
    //     gemm1.data_ptr<scalar_t>(), 
    //     gemm2.data_ptr<scalar_t>(),
    //     grad_output.data_ptr<scalar_t>(),
    //     size);
    // }));

    auto q_w = hadamard_weight / scale_weight;
    auto indicate_small = (q_w < -8).to(torch::kFloat16);
    auto indicate_big = (q_w > 7).to(torch::kFloat16);
    auto indicate_middle = 1.0 - indicate_small - indicate_big;
    auto grad_scale = 1.0 / sqrt(hadamard_weight.numel() * 7);
    auto grad_alpha = ((indicate_small * -8 + indicate_big * 7 + indicate_middle * (
                    -q_w + q_w.round())) * grad_output * grad_scale).sum().unsqueeze(0);
    auto grad_input = indicate_middle * grad_output; 
    //TODO:to test use this way, later change into Gumble

    // hipDeviceSynchronize();
    // clock_t time_dequantize_end = clock();

    // double quantize1_time = (double)(time_quantize1_end - time_quantize1_start) / CLOCKS_PER_SEC;
    // double quantize2_time = (double)(time_quantize2_end - time_quantize1_end) / CLOCKS_PER_SEC;
    // double leverage_time = (double)(time_leverage_end - time_quantize2_end) / CLOCKS_PER_SEC;
    // double sample_time = (double)(time_sample_end - time_leverage_end) / CLOCKS_PER_SEC;
    // double pack_time = (double)(time_pack_end - time_sample_end) / CLOCKS_PER_SEC;
    // double gemm1_time = (double)(time_gemm1_end - time_pack_end) / CLOCKS_PER_SEC;
    // double gemm2_time = (double)(time_gemm2_end - time_gemm1_end) / CLOCKS_PER_SEC;
    // double dequantize_time = (double)(time_dequantize_end - time_gemm2_end) / CLOCKS_PER_SEC;
    // // time_leverage_end

    // time_vector.push_back(quantize1_time);
    // time_vector.push_back(quantize2_time);
    // time_vector.push_back(leverage_time);
    // time_vector.push_back(sample_time);
    // time_vector.push_back(pack_time);
    // time_vector.push_back(gemm1_time);
    // time_vector.push_back(gemm2_time);
    // time_vector.push_back(dequantize_time);
    // auto sample_x = torch::cat({sample_x1, sample_x2}, 0);

    return std::make_tuple(grad_input, grad_alpha, grad_output);
    // return std::make_tuple(gemm1, gemm2, gemm3, gemm4, sum_y1_column, sum_y2_column);
}
